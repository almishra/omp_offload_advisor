
#include <hip/hip_runtime.h>
#include <stdio.h> 

int main() {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  if(nDevices < 1) {
    printf("No device found\n");
    return -1;
  }

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  printf("%d%d\n", prop.major, prop.minor);

  return 0;
}
